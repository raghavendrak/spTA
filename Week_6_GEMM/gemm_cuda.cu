#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>  // For atoi
#include <chrono>   // For timing

// Kernel to perform GEMM: C = A * B
__global__ void matrixMulKernel(double* A, double* B, double* C, int64_t m, int64_t n, int64_t k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < k) {
        double sum = 0.0;
        for (int i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

// Host function to set up kernel
void matrixMul(double* A, double* B, double* C, int64_t m, int64_t n, int64_t k, int thread_block_size) {
    double *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m * n * sizeof(double));
    hipMalloc((void**)&d_B, n * k * sizeof(double));
    hipMalloc((void**)&d_C, m * k * sizeof(double));
    
    hipMemcpy(d_A, A, m * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockDim(thread_block_size, thread_block_size);
    dim3 gridDim((k + thread_block_size - 1) / thread_block_size, (m + thread_block_size - 1) / thread_block_size);

    matrixMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, n, k);
    
    hipMemcpy(C, d_C, m * k * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(int argc, char* argv[]) {
    if (argc != 5) {
        std::cerr << "Usage: " << argv[0] << " <m> <n> <k> <thread_block_size>" << std::endl;
        return -1;
    }

    int64_t m = std::atoi(argv[1]);
    int64_t n = std::atoi(argv[2]);
    int64_t k = std::atoi(argv[3]);
    int thread_block_size = std::atoi(argv[4]);

    double* A = new double[m * n];
    double* B = new double[n * k];
    double* C_gpu = new double[m * k];

    for (int i = 0; i < m * n; i++) {
        A[i] = static_cast<double>(i + 1);
    }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < k; j++) {
            B[i * k + j] = (i == j) ? 1.0 : 0.0;
        }
    }

    // Measure GPU execution time
    auto start_gpu = std::chrono::high_resolution_clock::now();
    matrixMul(A, B, C_gpu, m, n, k, thread_block_size);
    auto end_gpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> gpu_time = end_gpu - start_gpu;

    std::cout << "Thread block size: " << thread_block_size << ", GPU execution time: " << gpu_time.count() << " seconds." << std::endl;

    delete[] A;
    delete[] B;
    delete[] C_gpu;

    return 0;
}
