#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstring>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "csf_tensor.h"
#include "matrix_utils.h"
#include <chrono>
using namespace std;

// Helper macro for checking CUDA errors
#define cudaCheckError(call)                                                        \
do {                                                                                 \
    hipError_t err = call;                                                          \
    if (err != hipSuccess) {                                                        \
        std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << " -> "        \
                  << hipGetErrorString(err) << " (" << err << ")" << std::endl;      \
        std::exit(EXIT_FAILURE);                                                     \
    }                                                                                \
} while (0)

/////////////////////////////////////////////////////////////////////
/*Start of device function for GPU 4 loop Method using warpshuffle and 2d grid*/
__global__ void GPU_4loop_ws(
  const uint64_t* __restrict__ mode_0_idx,
  const uint64_t* __restrict__ mode_1_ptr, const uint64_t* __restrict__ mode_1_idx,
  const uint64_t* __restrict__ mode_2_ptr, const uint64_t* __restrict__ mode_2_idx,
  const float* __restrict__ values, const float* __restrict__ arr_A, const float* __restrict__ arr_B,  
  float* arr_O, uint64_t f1, uint64_t f2, int ncm, uint64_t size_mode_0_idx
)
{
  extern __shared__ float buf[];
  uint64_t j, k, k_ptr, k_ptr_offset, index_A, index_B, index_O ;
  int r, s, r_offset, s_offset, WARP_SIZE = 32;
  float value, A_val;
  unsigned mask;

  // j_ptr = j_ptr_offset + blockIdx.x;
  // j = mode_1_idx[j_ptr];
  // // uint64_t nnz_k = mode_2_ptr[j_ptr+1] - mode_2_ptr[j_ptr];
  
  

  for(uint64_t i_ptr_offset = 0; i_ptr_offset < size_mode_0_idx; i_ptr_offset += gridDim.y){
    uint64_t i_ptr = i_ptr_offset + blockIdx.y;

    if(i_ptr < size_mode_0_idx){

      uint64_t i = mode_0_idx[i_ptr];
      for(uint64_t j_ptr_offset = mode_1_ptr[i_ptr]; j_ptr_offset < mode_1_ptr[i_ptr + 1]; j_ptr_offset += gridDim.x){
        uint64_t j_ptr = j_ptr_offset + blockIdx.x;

        if(j_ptr < mode_1_ptr[i_ptr + 1]){

          int buf_index = threadIdx.y * blockDim.x + threadIdx.x;

          //NOTE; WORKS ONLY IF f2 < 1024
          if(buf_index < f2){
            buf[buf_index] = 0.0;
          }
          __syncthreads();

          uint64_t j = mode_1_idx[j_ptr];
          // parallelize s across warps
          // block dimesion is 32 x 32. 
          // hence, each row of thread block will form a warp 
          // each row of thread block(a warp) picks a k, thus a nonzero of input tensor
          for(k_ptr_offset = mode_2_ptr[j_ptr]; k_ptr_offset < mode_2_ptr[j_ptr + 1]; k_ptr_offset += blockDim.x){
            k_ptr =  k_ptr_offset + threadIdx.x;
            if(k_ptr < mode_2_ptr[j_ptr + 1]){
              
              value = values[k_ptr];
              k = mode_2_idx[k_ptr];
              
              //Each thread in a warp picks a 's'
              for(s_offset = 0; s_offset < f2; s_offset += blockDim.y){ 
                s = s_offset + threadIdx.y;
                if(s < f2){
                  mask = __activemask();
                  index_B = k * f2 + s;
                  float prod_val = value * arr_B[index_B];
        
                  for(int shuffle_offset = WARP_SIZE/2; shuffle_offset > 0; shuffle_offset>>=1){
                    prod_val += __shfl_down_sync(mask, prod_val, shuffle_offset);
                  }
                  if(threadIdx.x == 0) buf[s] += prod_val;
                //   atomicAdd(&buf[s], value * arr_B[index_B] );
                }
              }
            }
          }
          __syncthreads();
          
          //////////////////////////////////////////////////////////////////////////////////
          // parallelize 'r' across warps
          // block dimesion is 32 x 32. 
          // hence, each row of thread block will form a warp 
          // each row of thread block(a warp) picks a 'r'
          if(ncm == 0){
            for(r_offset = 0; r_offset < f1; r_offset += blockDim.y){
              r = r_offset + threadIdx.y;
              if(r < f1){
                index_A = j * f1 + r;
                A_val = arr_A[index_A];
                //Each thread in a warp picks a 's'
                for(s_offset = 0; s_offset < f2; s_offset += blockDim.x){
                  s = s_offset + threadIdx.x;
                  if(s < f2){
                    index_O = i * f1 * f2 + r * f2  + s;
                    //atomic add is required since different threadblocks in the same stream has same i
                    atomicAdd(&arr_O[index_O], buf[s] * A_val);
                  }
                }
                
              }
            }
          }
          else if(ncm == 1){
            for(r_offset = 0; r_offset < f1; r_offset += blockDim.y){
              r = r_offset + threadIdx.y;
              if(r < f1){
                index_A = i * f1 + r;
                A_val = arr_A[index_A];
                //Each thread in a warp picks a 's'
                for(s_offset = 0; s_offset < f2; s_offset += blockDim.x){
                  s = s_offset + threadIdx.x;
                  if(s < f2){
                    index_O = j * f1 * f2 + r * f2  + s;
                    //atomic add is required since different threadblocks in the same stream has same i
                    atomicAdd(&arr_O[index_O], buf[s] * A_val);
                  }
                }
                
              }
            }
          }
        }
      }
    }
  }
}

__global__ void GPU_4loop_streams_ncm_2_part_1(
  // uint64_t* mode_1_ptr,
  uint64_t* mode_1_idx,
  uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
  float* values, float* arr_A, float* arr_B,  
  float* arr_O, uint64_t l, uint64_t m, uint64_t n, uint64_t f1, uint64_t f2, int ncm,
  int size_mode_0_ptr, int size_mode_1_ptr, int size_mode_2_ptr,
  int size_mode_0_idx, int size_mode_1_idx, int size_mode_2_idx, uint64_t i, uint64_t j_ptr_offset,
  float* buffer_for_ncm_2, bool* k_index_buffer
)
{ 
  //shared memory will not be enough for 2d dense buf[k,s] of type float
  // for e.g. dim_k = 1024, dim_s = 32, the required memory is 32*8*1024 = 256kb
  uint64_t j, j_ptr, k, k_ptr, k_ptr_offset, index_B ;
  int  s, s_offset, buf_index;// WARP_SIZE = 32;
  float value;
  // unsigned mask;

  j_ptr = j_ptr_offset + blockIdx.x;
  j = mode_1_idx[j_ptr];
  
  // parallelize s across warps
  // block dimesion is 32 x 32. 
  // hence, each row of thread block will form a warp 
  // each column of thread block(a warp) picks a k, thus a nonzero of input tensor
  for(k_ptr_offset = mode_2_ptr[j_ptr]; k_ptr_offset < mode_2_ptr[j_ptr + 1]; k_ptr_offset += blockDim.x){
    k_ptr =  k_ptr_offset + threadIdx.x;
    if(k_ptr < mode_2_ptr[j_ptr + 1]){
      
      value = values[k_ptr];
      k = mode_2_idx[k_ptr];
      //since each column in threadblock has same k, threads from first column is enough to note that k
      if(threadIdx.y == 0) k_index_buffer[k] = true;
      
      //Each thread in a warp picks a 's'
      for(s_offset = 0; s_offset < f2; s_offset += blockDim.y){
        s = s_offset + threadIdx.y;
        if(s < f2){
          index_B = j * f2 + s;
          buf_index = k * f2 + s;
          float prod_val = value * arr_B[index_B];
          
          //warp shuffle cannot be used here because either k or s is changing along the both block dimension
          // mask = __activemask();
          // for(int shuffle_offset = WARP_SIZE/2; shuffle_offset > 0; shuffle_offset>>=1){
          //   prod_val += __shfl_down_sync(mask, prod_val, shuffle_offset);
          // }
          // if(threadIdx.x == 0) 
          atomicAdd(&buffer_for_ncm_2[buf_index], prod_val);
        }
      }
    }
  }
  // __syncthreads(); won't work because synchronization across blocks is required 
  
  //////////////////////////////////////////////////////////////////////////////////
  
}



__global__ void GPU_4loop_streams_ncm_2_part_2(
  uint64_t* mode_1_idx,
  uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
  float* values, float* arr_A, float* arr_B,  
  float* arr_O, uint64_t l, uint64_t m, uint64_t n, uint64_t f1, uint64_t f2, int ncm,
  int size_mode_0_ptr, int size_mode_1_ptr, int size_mode_2_ptr,
  int size_mode_0_idx, int size_mode_1_idx, int size_mode_2_idx, uint64_t i, uint64_t j_ptr_offset,
  float* buffer_for_ncm_2, bool* k_index_buffer
)
{
  uint64_t  k,  index_A, index_O ;
  int r, s, r_offset, s_offset, buf_index;
  float  A_val;
  k = blockIdx.x;
  if(k_index_buffer[k]){
    // parallelize 'r' across warps
    // block dimesion is 32 x 32. 
    // hence, each row of thread block will form a warp 
    // each row of thread block(a warp) picks a 'r'

    // if(threadIdx.x == 0 && threadIdx.y == 0) printf("k = %d", k);
    for(r_offset = 0; r_offset < f1; r_offset += blockDim.y){
      r = r_offset + threadIdx.y;
      if(r < f1){
        index_A = i * f1 + r;
        A_val = arr_A[index_A];
        //Each thread in a warp picks a 's'
        for(s_offset = 0; s_offset < f2; s_offset += blockDim.x){
          s = s_offset + threadIdx.x;
          if(s < f2){
            index_O = k * f1 * f2 + r * f2  + s;
            buf_index = k * f2 + s;
            //atomic add is required since different threadblocks in the same stream has same i
            atomicAdd(&arr_O[index_O], buffer_for_ncm_2[buf_index] * A_val);
          }
        }
      }
    }
  }
}
/*End of device function for GPU 4 loop Method using warpshuffle and 2d grid*/
/////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////
/*Start of host function for GPU 4 loop Method using warpshuffle and 2d grid*/
void GPU_4loop_host_func(
  uint64_t* mode_0_ptr, uint64_t* mode_0_idx,
  uint64_t* mode_1_ptr, uint64_t* mode_1_idx,
  uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
  float* values, float* arr_A, float* arr_B,  
  float* arr_O, uint64_t arr_A_size, uint64_t arr_B_size, uint64_t arr_O_size, int contraction, 
  uint64_t l, uint64_t m, uint64_t n, uint64_t f1, uint64_t f2, uint64_t total_values,
  int size_mode_0_ptr, int size_mode_1_ptr, int size_mode_2_ptr,
  int size_mode_0_idx, int size_mode_1_idx, int size_mode_2_idx)
  {
    // Allocate device memory
    uint64_t *d_mode_0_idx, *d_mode_1_ptr;
    uint64_t *d_mode_1_idx, *d_mode_2_ptr, *d_mode_2_idx;
    float *d_values, *d_arr_A, *d_arr_B, *d_arr_O;
    // float* buffer_for_contraction_0_1;
    // float* buffer_for_contraction_2;
    // int* k_buffer_for_contraction_2;
  
    // hipMalloc(&d_mode_0_ptr, sizeof(uint64_t) * size_mode_0_ptr);
    hipMalloc(&d_mode_0_idx, sizeof(uint64_t) * size_mode_0_idx);
    hipMalloc(&d_mode_1_ptr, sizeof(uint64_t) * size_mode_1_ptr);
    hipMalloc(&d_mode_1_idx, sizeof(uint64_t) * size_mode_1_idx);
    hipMalloc(&d_mode_2_ptr, sizeof(uint64_t) * size_mode_2_ptr);
    hipMalloc(&d_mode_2_idx, sizeof(uint64_t) * size_mode_2_idx);
    hipMalloc(&d_values, sizeof(float) * total_values);
    hipMalloc(&d_arr_A, sizeof(float) * arr_A_size);
    hipMalloc(&d_arr_B, sizeof(float) * arr_B_size);
    hipMalloc(&d_arr_O, sizeof(float) * arr_O_size);
  
  
    // // parallelising 'j_ptr' for contraction = 0 and contraction = 1 :
    // hipMalloc(&buffer_for_contraction_0_1, f2 * size_mode_1_idx * sizeof(float));
  
    // // parallelising 'j_ptr' for contraction = 2 :
    // hipMalloc(&buffer_for_contraction_2, n * f2 * size_mode_1_idx * sizeof(float));
    // hipMalloc(&k_buffer_for_contraction_2, n * size_mode_1_idx * sizeof(int));
  
    // Copy data to device
    // hipMemcpy(d_mode_0_ptr, mode_0_ptr, sizeof(uint64_t) * size_mode_0_ptr, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_0_idx, mode_0_idx, sizeof(uint64_t) * size_mode_0_idx, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_1_ptr, mode_1_ptr, sizeof(uint64_t) * size_mode_1_ptr, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_1_idx, mode_1_idx, sizeof(uint64_t) * size_mode_1_idx, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_2_ptr, mode_2_ptr, sizeof(uint64_t) * size_mode_2_ptr, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_2_idx, mode_2_idx, sizeof(uint64_t) * size_mode_2_idx, hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, sizeof(float) * total_values, hipMemcpyHostToDevice);
    hipMemcpy(d_arr_A, arr_A, sizeof(float) * arr_A_size, hipMemcpyHostToDevice);
    hipMemcpy(d_arr_B, arr_B, sizeof(float) * arr_B_size, hipMemcpyHostToDevice);
    // hipMemcpy(d_arr_O, arr_O, sizeof(float) * arr_O_size, hipMemcpyHostToDevice);
    hipMemset(d_arr_O, 0, sizeof(float) * arr_O_size);
    
    
    // // parallelising 'j_ptr' for contraction = 0 and contraction = 1 :
    // hipMemset(buffer_for_contraction_0_1, 0, f2 * size_mode_1_idx * sizeof(float));
    
    
    // // parallelising 'j_ptr' for contraction = 2 :
    // hipMemset(buffer_for_contraction_2, 0, n * f2 * size_mode_1_idx * sizeof(float));
    // hipMemset(k_buffer_for_contraction_2, 0, n * size_mode_1_idx * sizeof(int));
    
    // uint64_t mode_1_idx_offset, mode_2_ptr_offset, mode_2_idx_offset, mode_1_idx_num_elements;
    // Launch kernels
    if (contraction == 0 || contraction == 1) {
      dim3 gridDim(32, 128);
      dim3 blockDim(32, 32);
      int sharedMemBytes = f2 * sizeof(float);

      auto start = std::chrono::high_resolution_clock::now();
      GPU_4loop_ws<<<gridDim, blockDim, sharedMemBytes>>>(
        d_mode_0_idx, d_mode_1_ptr, d_mode_1_idx, d_mode_2_ptr, d_mode_2_idx,
        d_values, d_arr_A, d_arr_B, d_arr_O, f1, f2, contraction, size_mode_0_idx
      );
      hipDeviceSynchronize();
      auto end = std::chrono::high_resolution_clock::now();
      auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
      cout << "Method: 2D-grid-2D-tb-ws-W, Time: " << duration / 1000.0 << " ms" << endl;
        
    }
    /*
    else if(contraction == 2){
      float* buffer_for_ncm_2;
      bool* k_index_buffer;
      
      NUM_STREAMS = 1;
      cout << "No. of streams = " << NUM_STREAMS <<endl;

      hipMalloc(&buffer_for_ncm_2, n * f2 * NUM_STREAMS * sizeof(float));
      hipMalloc(&k_index_buffer, n * NUM_STREAMS * sizeof(bool));
      
      // hipMalloc(&k_indices, n * NUM_STREAMS * sizeof(uint64_t));
      // hipMalloc(&counter,  NUM_STREAMS * sizeof(uint64_t));
      
      // hipMemset(buffer_for_ncm_2 , 0, n * f2  * NUM_STREAMS * sizeof(float));
      // hipMemset(k_index_buffer, 0, n  * NUM_STREAMS * sizeof(bool));

      
      for (uint64_t i_ptr = 0; i_ptr < mode_0_ptr[1]; ++i_ptr) {
        i = mode_0_idx[i_ptr];
        j_ptr_offset = mode_1_ptr[i_ptr];
        
        hipMemset(buffer_for_ncm_2 + n * f2 * (i_ptr % NUM_STREAMS), 0, n * f2  * sizeof(float));
        hipMemset(k_index_buffer + n * (i_ptr % NUM_STREAMS), 0, n  * sizeof(bool));
        
        dim3 gridDim(mode_1_ptr[i_ptr + 1] - mode_1_ptr[i_ptr]);
        dim3 blockDim(32, 32);

        GPU_4loop_streams_ncm_2_part_1<<<gridDim, blockDim, 0, streams[i_ptr%NUM_STREAMS]>>>(
          d_mode_1_idx, d_mode_2_ptr, d_mode_2_idx,
          d_values, d_arr_A, d_arr_B, d_arr_O, l, m, n, f1, f2, contraction,
          size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr,
          size_mode_0_idx, size_mode_1_idx, size_mode_2_idx,
          i, j_ptr_offset, buffer_for_ncm_2 + n * f2 * (i_ptr % NUM_STREAMS), k_index_buffer + n * (i_ptr % NUM_STREAMS)
        );

        // hipDeviceSynchronize();
        // pick_non_zero_Ks(k_index_buffer + n * (i_ptr % NUM_STREAMS), k_indices + n * (i_ptr % NUM_STREAMS),  n)

        gridDim.x = n; //TO-DO: have to be optimized
        GPU_4loop_streams_ncm_2_part_2<<<gridDim, blockDim, 0, streams[i_ptr%NUM_STREAMS]>>>(
          d_mode_1_idx, d_mode_2_ptr, d_mode_2_idx,
          d_values, d_arr_A, d_arr_B, d_arr_O, l, m, n, f1, f2, contraction,
          size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr,
          size_mode_0_idx, size_mode_1_idx, size_mode_2_idx,
          i, j_ptr_offset, buffer_for_ncm_2 + n * (i_ptr % NUM_STREAMS), k_index_buffer + n * (i_ptr % NUM_STREAMS)
        );
        hipGetLastError();  // Check launch err;
        // hipStreamSynchronize(streams[i_ptr % NUM_STREAMS]);
      }
      
    }
    */

    hipDeviceSynchronize();
    // Copy results back to host
    hipMemcpy(arr_O, d_arr_O, sizeof(float) * arr_O_size, hipMemcpyDeviceToHost);
  
    // Free device memory
    // hipFree(d_mode_0_ptr);
    hipFree(d_mode_0_idx);
    hipFree(d_mode_1_ptr);
    hipFree(d_mode_1_idx);
    hipFree(d_mode_2_ptr);
    hipFree(d_mode_2_idx);
    hipFree(d_values);
    hipFree(d_arr_A);
    hipFree(d_arr_B);
    hipFree(d_arr_O);
  
    // hipFree(buffer_for_contraction_0_1);
    // hipFree(buffer_for_contraction_2);
    // hipFree(k_buffer_for_contraction_2);
  }
/*End of host function for GPU 4 loop Method using warpshuffle and 2d grid*/
////////////////////////////////////////////////////////////////////

// Include the reference implementation for validation
#define INCLUDED_AS_LIBRARY
#include "v2_cpu_4loop.cu"

int main(int argc, char* argv[]) {
    bool verbose = false;
    string csf_file;
    uint64_t rank1 = 30, rank2 = 30;
    int ncm = 0;
    bool verify = false;  // Default: don't verify results
    
    // Parse command line arguments
    for (int i = 1; i < argc; i++) {
        string arg = argv[i];
        if (arg == "-v" || arg == "--verbose") {
            verbose = true;
        } else if (arg == "-r1" && i + 1 < argc) {
            rank1 = atoi(argv[++i]);
        } else if (arg == "-r2" && i + 1 < argc) {
            rank2 = atoi(argv[++i]);
        } else if ((arg == "-n" || arg == "--ncm") && i + 1 < argc) {
            ncm = atoi(argv[++i]);
        } else if (arg == "--verify") {
            verify = true;
        } else if (csf_file.empty()) {
            csf_file = arg;
        }
    }
    
    if (csf_file.empty()) {
        cerr << "Usage: " << argv[0] << " [options] <csf_file>" << endl;
        cerr << "Options:" << endl;
        cerr << "  -v, --verbose      Enable verbose output" << endl;
        cerr << "  -r1 <rank>         Set first factor matrix rank (default 30)" << endl;
        cerr << "  -r2 <rank>         Set second factor matrix rank (default 30)" << endl;
        cerr << "  -n, --ncm <mode>   Set contraction mode (0, 1, or 2, default 0)" << endl;
        cerr << "  --verify           Verify results against reference implementation" << endl;
        return 1;
    }
    
    try {
        // Load the CSF tensor
        CSFTensor tensor = readCSFTensor(csf_file);
        
        if (verbose) {
            cout << "Loaded tensor from " << csf_file << endl;
            cout << "Tensor dimensions: " << tensor.dimensions[0] << " x " << tensor.dimensions[1] << " x " << tensor.dimensions[2] << endl;
            cout << "Nonzeros: " << tensor.values.size() << endl;
        }
        
        // Convert CSF tensor to arrays
        uint64_t *mode_0_ptr, *mode_0_idx;
        uint64_t *mode_1_ptr, *mode_1_idx;
        uint64_t *mode_2_ptr, *mode_2_idx;
        float *values;
        int order;
        
        size_t size_mode_0_ptr = tensor.ptrs[0].size();
        size_t size_mode_1_ptr = tensor.ptrs[1].size();
        size_t size_mode_2_ptr = tensor.ptrs[2].size();
        size_t size_mode_0_idx = tensor.idxs[0].size();
        size_t size_mode_1_idx = tensor.idxs[1].size();
        size_t size_mode_2_idx = tensor.idxs[2].size();
        size_t total_values = tensor.values.size();
        
        vector<uint64_t> dimensions(tensor.order);
        for(int i = 0; i < tensor.order; i++){
            dimensions[i] = tensor.dimensions[i];
        }

        getCSFArrays(tensor, &mode_0_ptr, &mode_0_idx, 
                    &mode_1_ptr, &mode_1_idx, 
                    &mode_2_ptr, &mode_2_idx, 
                    &values, &order);
        
        // Calculate matrix dimensions based on contraction mode
        uint64_t matrix_dim1 = getMatrixDim1(dimensions, ncm);
        uint64_t matrix_dim2 = getMatrixDim2(dimensions, ncm);
        uint64_t out_dim1 = getOutputDim1(dimensions, ncm);
        
        // Generate factor matrices
        float *arr_A = nullptr, *arr_B = nullptr;
        generate_matrix(matrix_dim1, rank1, 42, arr_A);
        generate_matrix(matrix_dim2, rank2, 43, arr_B);
        
        // Prepare output matrix dimensions
        uint64_t out_dim2 = rank1 * rank2;
        uint64_t arr_A_size = matrix_dim1 * rank1;
        uint64_t arr_B_size = matrix_dim2 * rank2;
        uint64_t arr_O_size = out_dim1 * out_dim2;
        
        if (verbose) {
            cout << "Matrix A dimensions: " << matrix_dim1 << " x " << rank1 << endl;
            cout << "Matrix B dimensions: " << matrix_dim2 << " x " << rank2 << endl;
            cout << "Output dimensions: " << out_dim1 << " x " << out_dim2 << endl;
        }
        
        // Allocate output array
        float* arr_O = allocate_aligned_array(arr_O_size);
        float* ref_O = nullptr;
        
        if (verify) {
            // Only allocate reference array if verification is needed
            ref_O = allocate_aligned_array(arr_O_size);
        }
        
        // Run this implementation (GPU 4-loop with streams) first
        if (verbose) {
            cout << "Running GPU 4-loop with streams implementation..." << endl;
        }
        auto start = std::chrono::high_resolution_clock::now();
        
        // pinned memory for streams
        size_t ptr_size_0 = sizeof(uint64_t) * size_mode_0_ptr;
        size_t idx_size_0 = sizeof(uint64_t) * size_mode_0_idx;
        size_t ptr_size_1 = sizeof(uint64_t) * size_mode_1_ptr;
        size_t idx_size_1 = sizeof(uint64_t) * size_mode_1_idx;
        size_t ptr_size_2 = sizeof(uint64_t) * size_mode_2_ptr;
        size_t idx_size_2 = sizeof(uint64_t) * size_mode_2_idx;
        size_t val_size   = sizeof(float)   * total_values;

        // Register host memory
        hipHostRegister(mode_0_ptr, ptr_size_0, hipHostRegisterDefault);
        hipHostRegister(mode_0_idx, idx_size_0, hipHostRegisterDefault);
        hipHostRegister(mode_1_ptr, ptr_size_1, hipHostRegisterDefault);
        hipHostRegister(mode_1_idx, idx_size_1, hipHostRegisterDefault);
        hipHostRegister(mode_2_ptr, ptr_size_2, hipHostRegisterDefault);
        hipHostRegister(mode_2_idx, idx_size_2, hipHostRegisterDefault);
        hipHostRegister(values,     val_size,   hipHostRegisterDefault);

        GPU_4loop_host_func(
            mode_0_ptr, mode_0_idx,
            mode_1_ptr, mode_1_idx,
            mode_2_ptr, mode_2_idx,
            values, arr_A, arr_B, arr_O,
            arr_A_size, arr_B_size, arr_O_size,
            ncm, dimensions[0], dimensions[1], dimensions[2], rank1, rank2,
            total_values,
            size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr,
            size_mode_0_idx, size_mode_1_idx, size_mode_2_idx
        );
        
        hipHostUnregister(mode_0_ptr);
        hipHostUnregister(mode_0_idx);
        hipHostUnregister(mode_1_ptr);
        hipHostUnregister(mode_1_idx);
        hipHostUnregister(mode_2_ptr);
        hipHostUnregister(mode_2_idx);
        hipHostUnregister(values);
        
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
        
        bool valid = true;
        float ref_duration = 0.0;
        
        if (verify) {
            // Only run reference implementation and validate if requested
            if (verbose) {
                cout << "Running reference implementation (CPU 4-loop)..." << endl;
            }
            auto ref_start = std::chrono::high_resolution_clock::now();
            
            performContraction_cpu_2(
                mode_0_ptr, mode_0_idx,
                mode_1_ptr, mode_1_idx,
                mode_2_ptr, mode_2_idx,
                values, arr_A, arr_B, ref_O,
                arr_A_size, arr_B_size, arr_O_size, ncm,
                dimensions[0], dimensions[1], dimensions[2], rank1, rank2
            );
            
            auto ref_end = std::chrono::high_resolution_clock::now();
            ref_duration = std::chrono::duration_cast<std::chrono::microseconds>(ref_end - ref_start).count();
            
            // Validate results using compare_results from matrix_utils.h
            valid = compare_results(arr_O, ref_O, arr_O_size);
            cout << "Result validation: " << (valid ? "PASSED" : "FAILED") << endl;
        }
        
        // Report results
        if(verbose){  
          cout << "Method: GPU_4L_WS2, Time: " << duration / 1000.0 << " ms" << endl;
        }
        
        // Clean up
        delete[] mode_0_ptr;
        delete[] mode_0_idx;
        delete[] mode_1_ptr;
        delete[] mode_1_idx;
        delete[] mode_2_ptr;
        delete[] mode_2_idx;
        delete[] values;
        delete[] arr_A;
        delete[] arr_B;
        free(arr_O);
        if (ref_O) free(ref_O);
        
        return valid ? 0 : 1;
    }
    catch (const std::exception& e) {
        cerr << "Error: " << e.what() << endl;
        return 1;
    }
}