#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstring>
#include <stdexcept>
#include <chrono>
#include <hip/hip_runtime.h>
#include "csf_tensor.h"
#include "matrix_utils.h"

using namespace std;

// Helper macro for checking CUDA errors
#define cudaCheckError(call)                                                        \
do {                                                                                 \
    hipError_t err = call;                                                          \
    if (err != hipSuccess) {                                                        \
        std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << " -> "        \
                  << hipGetErrorString(err) << " (" << err << ")" << std::endl;      \
        std::exit(EXIT_FAILURE);                                                     \
    }                                                                                \
} while (0)



/////////////////////////////////////////////////////////////////////
/*Start of device function for GPU 5 loop Method*/
__global__ void GPU_5loop_contraction_kernel_0(
  uint64_t* mode_0_ptr, uint64_t* mode_0_idx,
  uint64_t* mode_1_ptr, uint64_t* mode_1_idx,
  uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
  Scalar* values, Scalar* arr_A, Scalar* arr_B,  
  Scalar* arr_O, uint64_t l, uint64_t m, uint64_t n, uint64_t f1, uint64_t f2, int contraction,
  uint64_t size_mode_0_ptr, uint64_t size_mode_1_ptr, uint64_t size_mode_2_ptr,
  uint64_t size_mode_0_idx, uint64_t size_mode_1_idx, uint64_t size_mode_2_idx) 
{
  // Compute thread index
  uint64_t j_ptr = blockIdx.x * blockDim.x + threadIdx.x;

  // Find the i_ptr associated with the current j_ptr
  int64_t i_ptr = -1;
  for (uint64_t p = 0; p < size_mode_1_ptr - 1; ++p) {
    if (mode_1_ptr[p] <= j_ptr && j_ptr < mode_1_ptr[p + 1]) {
      i_ptr = p;
      break;
    }
  }
  uint64_t i, j, k, index_A, index_B, index_O;
  Scalar value;
  if ((i_ptr >= 0 && i_ptr < mode_0_ptr[1]) && 
      ( j_ptr < size_mode_1_idx) ) 
  {
    for (uint64_t k_ptr = mode_2_ptr[j_ptr]; k_ptr < mode_2_ptr[j_ptr + 1]; ++k_ptr) {
    
      i = mode_0_idx[i_ptr] ;
      j = mode_1_idx[j_ptr] ;

      k = mode_2_idx[k_ptr] ;
      value = values[k_ptr];

      for (uint64_t r = 0; r < f1; ++r) {
        index_A = 0;
        if (contraction == 0) {
          index_A = j * f1 + r;
        } else if (contraction == 1) {
          index_A = i * f1 + r;
        } else if (contraction == 2) {
          index_A = i * f1 + r;
        }

        for (uint64_t s = 0; s < f2; ++s) {
          if (contraction == 0) {
            index_B = k * f2 + s;
            index_O = i * f1 * f2 + r * f2 + s;
          } else if (contraction == 1) {
            index_B = k * f2 + s;
            index_O = j * f1 * f2 + r * f2 + s;
          } else if (contraction == 2) {
            index_B = j * f2 + s;
            index_O = k * f1 * f2 + r * f2 + s;
          }

          atomicAdd(&arr_O[index_O], value * arr_A[index_A] * arr_B[index_B]);
        }
      }
    }
  }
}

/*End of device function for GPU 5 loop Method*/
/////////////////////////////////////////////////////////////////////

/////////////////////////////////////////////////////////////////////
/*Start of host function for GPU 5 loop Method*/

void performContraction_gpu_1(
  uint64_t* mode_0_ptr, uint64_t* mode_0_idx,
  uint64_t* mode_1_ptr, uint64_t* mode_1_idx,
  uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
  Scalar* values, Scalar* arr_A, Scalar* arr_B,
  Scalar* arr_O, uint64_t arr_A_size, uint64_t arr_B_size, uint64_t arr_O_size,
  int contraction, uint64_t dim_0, uint64_t dim_1, uint64_t dim_2,
  uint64_t r1, uint64_t r2, uint64_t total_values,
  uint64_t size_mode_0_ptr, uint64_t size_mode_1_ptr, uint64_t size_mode_2_ptr,
  uint64_t size_mode_0_idx, uint64_t size_mode_1_idx, uint64_t size_mode_2_idx)
{
  // Allocate device memory
  uint64_t *d_mode_0_ptr, *d_mode_0_idx, *d_mode_1_ptr, *d_mode_1_idx, *d_mode_2_ptr, *d_mode_2_idx;
  Scalar *d_values, *d_arr_A, *d_arr_B, *d_arr_O;

  hipMalloc(&d_mode_0_ptr, sizeof(uint64_t) * size_mode_0_ptr);
  hipMalloc(&d_mode_0_idx, sizeof(uint64_t) * size_mode_0_idx);
  hipMalloc(&d_mode_1_ptr, sizeof(uint64_t) * size_mode_1_ptr);
  hipMalloc(&d_mode_1_idx, sizeof(uint64_t) * size_mode_1_idx);
  hipMalloc(&d_mode_2_ptr, sizeof(uint64_t) * size_mode_2_ptr);
  hipMalloc(&d_mode_2_idx, sizeof(uint64_t) * size_mode_2_idx);
  hipMalloc(&d_values, sizeof(Scalar) * total_values);
  hipMalloc(&d_arr_A, sizeof(Scalar) * arr_A_size);
  hipMalloc(&d_arr_B, sizeof(Scalar) * arr_B_size);
  hipMalloc(&d_arr_O, sizeof(Scalar) * arr_O_size);

  // Copy data from host to device
  hipMemcpy(d_mode_0_ptr, mode_0_ptr, sizeof(uint64_t) * size_mode_0_ptr, hipMemcpyHostToDevice);
  hipMemcpy(d_mode_0_idx, mode_0_idx, sizeof(uint64_t) * size_mode_0_idx, hipMemcpyHostToDevice);
  hipMemcpy(d_mode_1_ptr, mode_1_ptr, sizeof(uint64_t) * size_mode_1_ptr, hipMemcpyHostToDevice);
  hipMemcpy(d_mode_1_idx, mode_1_idx, sizeof(uint64_t) * size_mode_1_idx, hipMemcpyHostToDevice);
  hipMemcpy(d_mode_2_ptr, mode_2_ptr, sizeof(uint64_t) * size_mode_2_ptr, hipMemcpyHostToDevice);
  hipMemcpy(d_mode_2_idx, mode_2_idx, sizeof(uint64_t) * size_mode_2_idx, hipMemcpyHostToDevice);
  hipMemcpy(d_values, values, sizeof(Scalar) * total_values, hipMemcpyHostToDevice);
  hipMemcpy(d_arr_A, arr_A, sizeof(Scalar) * arr_A_size, hipMemcpyHostToDevice);
  hipMemcpy(d_arr_B, arr_B, sizeof(Scalar) * arr_B_size, hipMemcpyHostToDevice);
  hipMemset(d_arr_O, 0, sizeof(Scalar) * arr_O_size);

  // Kernel launch parameters
  int threadsPerBlock = 256;
  int blocksPerGrid = (size_mode_1_idx + threadsPerBlock - 1) / threadsPerBlock;

  // Launch appropriate kernel based on contraction type
  GPU_5loop_contraction_kernel_0<<<blocksPerGrid, threadsPerBlock>>>(
    d_mode_0_ptr, d_mode_0_idx, d_mode_1_ptr, d_mode_1_idx, d_mode_2_ptr, d_mode_2_idx,
    d_values, d_arr_A, d_arr_B, d_arr_O, dim_0, dim_1, dim_2, r1, r2, contraction,
    size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr, size_mode_0_idx, size_mode_1_idx, size_mode_2_idx
  );

  // Check for launch errors
  hipGetLastError();
  hipDeviceSynchronize();

  // Copy results back to host
  hipMemcpy(arr_O, d_arr_O, sizeof(Scalar) * arr_O_size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_mode_0_ptr);
  hipFree(d_mode_0_idx);
  hipFree(d_mode_1_ptr);
  hipFree(d_mode_1_idx);
  hipFree(d_mode_2_ptr);
  hipFree(d_mode_2_idx);
  hipFree(d_values);
  hipFree(d_arr_A);
  hipFree(d_arr_B);
  hipFree(d_arr_O);
}

/*End of host function for GPU 5 loop Method*/
////////////////////////////////////////////////////////////////////

// Include the reference implementation for validation
#define INCLUDED_AS_LIBRARY
#include "v2_cpu_4loop.cu"

int main(int argc, char* argv[]) {
    bool verbose = false;
    string csf_file;
    uint64_t rank1 = 30, rank2 = 30;
    int ncm = 0;
    bool verify = false;  // Default: don't verify results
    
    // Parse command line arguments
    for (int i = 1; i < argc; i++) {
        string arg = argv[i];
        if (arg == "-v" || arg == "--verbose") {
            verbose = true;
        } else if (arg == "-r1" && i + 1 < argc) {
            rank1 = atoi(argv[++i]);
        } else if (arg == "-r2" && i + 1 < argc) {
            rank2 = atoi(argv[++i]);
        } else if ((arg == "-n" || arg == "--ncm") && i + 1 < argc) {
            ncm = atoi(argv[++i]);
        } else if (arg == "--verify") {
            verify = true;
        } else if (csf_file.empty()) {
            csf_file = arg;
        }
    }
    
    if (csf_file.empty()) {
        cerr << "Usage: " << argv[0] << " [options] <csf_file>" << endl;
        cerr << "Options:" << endl;
        cerr << "  -v, --verbose      Enable verbose output" << endl;
        cerr << "  -r1 <rank>         Set first factor matrix rank (default 30)" << endl;
        cerr << "  -r2 <rank>         Set second factor matrix rank (default 30)" << endl;
        cerr << "  -n, --ncm <mode>   Set contraction mode (0, 1, or 2, default 0)" << endl;
        cerr << "  --verify           Verify results against reference implementation" << endl;
        return 1;
    }
    
    try {
        // Load the CSF tensor
        CSFTensor tensor = readCSFTensor(csf_file);
        
        if (verbose) {
            cout << "Loaded tensor from " << csf_file << endl;
            cout << "Tensor dimensions: " << tensor.dimensions[0] << " x " << tensor.dimensions[1] << " x " << tensor.dimensions[2] << endl;
            cout << "Nonzeros: " << tensor.values.size() << endl;
        }
        
        // Convert CSF tensor to arrays
        uint64_t *mode_0_ptr, *mode_0_idx;
        uint64_t *mode_1_ptr, *mode_1_idx;
        uint64_t *mode_2_ptr, *mode_2_idx;
        Scalar *values;
        int order;
        
        size_t size_mode_0_ptr = tensor.ptrs[0].size();
        size_t size_mode_1_ptr = tensor.ptrs[1].size();
        size_t size_mode_2_ptr = tensor.ptrs[2].size();
        size_t size_mode_0_idx = tensor.idxs[0].size();
        size_t size_mode_1_idx = tensor.idxs[1].size();
        size_t size_mode_2_idx = tensor.idxs[2].size();
        size_t total_values = tensor.values.size();
        
        vector<uint64_t> dimensions(tensor.order);
        for(int i = 0; i < tensor.order; i++){
            dimensions[i] = tensor.dimensions[i];
        }

        getCSFArrays(tensor, &mode_0_ptr, &mode_0_idx, 
                    &mode_1_ptr, &mode_1_idx, 
                    &mode_2_ptr, &mode_2_idx, 
                    &values, &order);
        
        // Calculate matrix dimensions based on contraction mode
        uint64_t matrix_dim1 = getMatrixDim1(dimensions, ncm);
        uint64_t matrix_dim2 = getMatrixDim2(dimensions, ncm);
        uint64_t out_dim1 = getOutputDim1(dimensions, ncm);
        
        // Generate factor matrices
        Scalar *arr_A = nullptr, *arr_B = nullptr;
        generate_matrix(matrix_dim1, rank1, 42, arr_A);
        generate_matrix(matrix_dim2, rank2, 43, arr_B);
        
        // Prepare output matrix dimensions
        uint64_t out_dim2 = rank1 * rank2;
        uint64_t arr_A_size = matrix_dim1 * rank1;
        uint64_t arr_B_size = matrix_dim2 * rank2;
        uint64_t arr_O_size = out_dim1 * out_dim2;
        
        if (verbose) {
            cout << "Matrix A dimensions: " << matrix_dim1 << " x " << rank1 << endl;
            cout << "Matrix B dimensions: " << matrix_dim2 << " x " << rank2 << endl;
            cout << "Output dimensions: " << out_dim1 << " x " << out_dim2 << endl;
        }
        
        // Allocate output array
        Scalar* arr_O = allocate_aligned_array(arr_O_size);
        Scalar* ref_O = nullptr;
        
        if (verify) {
            // Only allocate reference array if verification is needed
            ref_O = allocate_aligned_array(arr_O_size);
        }
        
        // Run this implementation (GPU 5-loop) first
        if (verbose) {
            cout << "Running GPU 5-loop implementation..." << endl;
        }
        auto start = std::chrono::high_resolution_clock::now();
        
        performContraction_gpu_1(
            mode_0_ptr, mode_0_idx,
            mode_1_ptr, mode_1_idx,
            mode_2_ptr, mode_2_idx,
            values, arr_A, arr_B, arr_O,
            arr_A_size, arr_B_size, arr_O_size,
            ncm, dimensions[0], dimensions[1], dimensions[2], rank1, rank2,
            total_values,
            size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr,
            size_mode_0_idx, size_mode_1_idx, size_mode_2_idx
        );
        
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
        
        bool valid = true;
        Scalar ref_duration = 0.0;
        
        if (verify) {
            // Now run reference implementation (CPU 4-loop) for validation
            if (verbose) {
                cout << "Running reference implementation (CPU 4-loop)..." << endl;
            }
            auto ref_start = std::chrono::high_resolution_clock::now();
            
            performContraction_cpu_2(
                mode_0_ptr, mode_0_idx,
                mode_1_ptr, mode_1_idx,
                mode_2_ptr, mode_2_idx,
                values, arr_A, arr_B, ref_O,
                arr_A_size, arr_B_size, arr_O_size, ncm,
                dimensions[0], dimensions[1], dimensions[2], rank1, rank2
            );
            
            auto ref_end = std::chrono::high_resolution_clock::now();
            ref_duration = std::chrono::duration_cast<std::chrono::microseconds>(ref_end - ref_start).count();
            
            // Validate results using compare_results from matrix_utils.h
            valid = compare_results(arr_O, ref_O, arr_O_size);
        }
        
        // Report results
        if (verbose) {
            cout << "GPU 5-loop execution time: " << duration / 1000.0 << " ms" << endl;
            if (verify) {
                cout << "Reference execution time: " << ref_duration / 1000.0 << " ms" << endl;
                cout << "Speedup over reference: " << (Scalar)ref_duration / duration << "x" << endl;
                cout << "Result validation: " << (valid ? "PASSED" : "FAILED") << endl;
            }
        } else {
            if (verify) {
                cout << "Method: GPU_5L, Time: " << duration / 1000.0 << " ms, Validation: " << (valid ? "PASSED" : "FAILED") << endl;
            } else {
                cout << "Method: GPU_5L, Time: " << duration / 1000.0 << " ms" << endl;
            }
        }
        
        // Clean up
        delete[] mode_0_ptr;
        delete[] mode_0_idx;
        delete[] mode_1_ptr;
        delete[] mode_1_idx;
        delete[] mode_2_ptr;
        delete[] mode_2_idx;
        delete[] values;
        delete[] arr_A;
        delete[] arr_B;
        free(arr_O);
        if (ref_O) free(ref_O);
        
        return valid ? 0 : 1;
    }
    catch (const std::exception& e) {
        cerr << "Error: " << e.what() << endl;
        return 1;
    }
}
