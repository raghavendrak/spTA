#include "hip/hip_runtime.h"
//TO-DO: write kernel for contraction = 2 i.e. ijk, ir, js -> rsk
#include <iostream>
#include <vector>
#include <cstring>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "csf_tensor.h"
#include "matrix_utils.h"

using namespace std;

// Helper macro for checking CUDA errors
#define cudaCheckError(call)                                                        \
do {                                                                                 \
    hipError_t err = call;                                                          \
    if (err != hipSuccess) {                                                        \
        std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << " -> "        \
                  << hipGetErrorString(err) << " (" << err << ")" << std::endl;      \
        std::exit(EXIT_FAILURE);                                                     \
    }                                                                                \
} while (0)

/////////////////////////////////////////////////////////////////////
/*Start of device function for GPU 4 loop Method - each thread picks a k */
__global__ void GPU_4loop_parK(
uint64_t* mode_0_ptr, uint64_t* mode_0_idx,
uint64_t* mode_1_ptr, uint64_t* mode_1_idx,
uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
float* values, float* arr_A, float* arr_B,  
float* arr_O, uint64_t l, uint64_t m, uint64_t n, uint64_t f1, uint64_t f2, int contraction,
uint64_t size_mode_0_ptr, uint64_t size_mode_1_ptr, uint64_t size_mode_2_ptr,
uint64_t size_mode_0_idx, uint64_t size_mode_1_idx, uint64_t size_mode_2_idx, float* buffer) 
{ 
  
  // extern __shared__ float buf[];
  // Compute thread index
  int64_t k_ptr = blockIdx.x * blockDim.x + threadIdx.x;
  if(k_ptr >= size_mode_2_idx) return;

  // Find the j_ptr associated with the current k_ptr
  int64_t j_ptr = -1;
  for (uint64_t p = 0; p < size_mode_2_ptr - 1; ++p) {
    if (mode_2_ptr[p] <= k_ptr && k_ptr < mode_2_ptr[p + 1]) {
      j_ptr = p;
      break;
    }
  }

  // Find the i_ptr associated with the current j_ptr
  int64_t i_ptr = -1;
  for (uint64_t p = 0; p < size_mode_1_ptr - 1; ++p) {
    if (mode_1_ptr[p] <= j_ptr && j_ptr < mode_1_ptr[p + 1]) {
      i_ptr = p;
      break;
    }
  }

  uint64_t i, j, k;
  float value;
  if ((i_ptr >= 0 && i_ptr < mode_0_ptr[1]) && 
      (j_ptr >= 0 && j_ptr < size_mode_1_idx) && 
      (k_ptr >= 0 && k_ptr < size_mode_2_idx)) 
  {
    i = mode_0_idx[i_ptr] ;
    j = mode_1_idx[j_ptr] ;
    k = mode_2_idx[k_ptr] ;
    value = values[k_ptr];

    for (uint64_t s = 0; s < f2; s++) {
      atomicAdd(&buffer[ j_ptr * f2 + s], value * arr_B[k * f2 + s]);  
    }
  }

}
__global__ void GPU_4loop_parJ(
uint64_t* mode_0_ptr, uint64_t* mode_0_idx,
uint64_t* mode_1_ptr, uint64_t* mode_1_idx,
uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
float* values, float* arr_A, float* arr_B,  
float* arr_O, uint64_t l, uint64_t m, uint64_t n, uint64_t f1, uint64_t f2, int contraction,
uint64_t size_mode_0_ptr, uint64_t size_mode_1_ptr, uint64_t size_mode_2_ptr,
uint64_t size_mode_0_idx, uint64_t size_mode_1_idx, uint64_t size_mode_2_idx, float* buffer) 
{ 
  
  // extern __shared__ float buf[];
  // Compute thread index
  int64_t j_ptr = blockIdx.x * blockDim.x + threadIdx.x;
  if(j_ptr >= size_mode_1_idx) return;

  // Find the i_ptr associated with the current j_ptr
  int64_t i_ptr = -1;
  for (uint64_t p = 0; p < size_mode_1_ptr - 1; ++p) {
    if (mode_1_ptr[p] <= j_ptr && j_ptr < mode_1_ptr[p + 1]) {
      i_ptr = p;
      break;
    }
  }

  uint64_t i, j, index_A,  index_O;
  if ((i_ptr >= 0 && i_ptr < mode_0_ptr[1]) && 
      (j_ptr >= 0 && j_ptr < size_mode_1_idx)) 
  {
    i = mode_0_idx[i_ptr] ;
    j = mode_1_idx[j_ptr] ;
    
    for (uint64_t r = 0; r < f1; ++r) {
      uint64_t index_A = 0;
      if (contraction == 0){
        index_A = j * f1 + r;
      }
      else if(contraction == 1){
        index_A = i * f1 + r;
      }

      for (uint64_t s = 0; s < f2; ++s) {
        uint64_t index_O = 0;
        if (contraction == 0) {
          index_O = i * f1 * f2 + r * f2 + s;
        } else if (contraction == 1) {
          index_O = j * f1 * f2 + r * f2 + s;
        }
        atomicAdd(&arr_O[index_O], buffer[j_ptr * f2 + s] * arr_A[index_A]);
      }
    }
    
  }

}

/*End of device function for GPU 4 loop Method using STREAMS*/
/////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////
/*Start of host function for GPU 4 loop Method using parK*/
void GPU_4loop_parK_host_func(
  uint64_t* mode_0_ptr, uint64_t* mode_0_idx,
  uint64_t* mode_1_ptr, uint64_t* mode_1_idx,
  uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
  float* values, float* arr_A, float* arr_B,  
  float* arr_O, uint64_t arr_A_size, uint64_t arr_B_size, uint64_t arr_O_size, int contraction, 
  uint64_t l, uint64_t m, uint64_t n, uint64_t f1, uint64_t f2, uint64_t total_values,
  int size_mode_0_ptr, int size_mode_1_ptr, int size_mode_2_ptr,
  int size_mode_0_idx, int size_mode_1_idx, int size_mode_2_idx)
  {
    // Allocate device memory
    uint64_t *d_mode_0_ptr, *d_mode_0_idx, *d_mode_1_ptr, *d_mode_1_idx, *d_mode_2_ptr, *d_mode_2_idx;
    float *d_values, *d_arr_A, *d_arr_B, *d_arr_O;
   
  
    hipMalloc(&d_mode_0_ptr, sizeof(uint64_t) * size_mode_0_ptr);
    hipMalloc(&d_mode_0_idx, sizeof(uint64_t) * size_mode_0_idx);
    hipMalloc(&d_mode_1_ptr, sizeof(uint64_t) * size_mode_1_ptr);
    hipMalloc(&d_mode_1_idx, sizeof(uint64_t) * size_mode_1_idx);
    hipMalloc(&d_mode_2_ptr, sizeof(uint64_t) * size_mode_2_ptr);
    hipMalloc(&d_mode_2_idx, sizeof(uint64_t) * size_mode_2_idx);
    hipMalloc(&d_values, sizeof(float) * total_values);
    hipMalloc(&d_arr_A, sizeof(float) * arr_A_size);
    hipMalloc(&d_arr_B, sizeof(float) * arr_B_size);
    hipMalloc(&d_arr_O, sizeof(float) * arr_O_size);
  
  
    // Copy data to device
    hipMemcpy(d_mode_0_ptr, mode_0_ptr, sizeof(uint64_t) * size_mode_0_ptr, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_0_idx, mode_0_idx, sizeof(uint64_t) * size_mode_0_idx, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_1_ptr, mode_1_ptr, sizeof(uint64_t) * size_mode_1_ptr, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_1_idx, mode_1_idx, sizeof(uint64_t) * size_mode_1_idx, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_2_ptr, mode_2_ptr, sizeof(uint64_t) * size_mode_2_ptr, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_2_idx, mode_2_idx, sizeof(uint64_t) * size_mode_2_idx, hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, sizeof(float) * total_values, hipMemcpyHostToDevice);
    hipMemcpy(d_arr_A, arr_A, sizeof(float) * arr_A_size, hipMemcpyHostToDevice);
    hipMemcpy(d_arr_B, arr_B, sizeof(float) * arr_B_size, hipMemcpyHostToDevice);
    hipMemset(d_arr_O, 0, sizeof(float) * arr_O_size);
    
    
    
    if (contraction == 0 || contraction == 1) {

      float* buffer_for_contraction_0_1;
      cudaCheckError(hipMalloc(&buffer_for_contraction_0_1, f2 * size_mode_1_idx * sizeof(float)));
      cudaCheckError(hipMemset(buffer_for_contraction_0_1, 0, f2 * size_mode_1_idx * sizeof(float)));
      
      int threadsPerBlock = 256;
      int blocksPerGrid = (size_mode_2_idx + threadsPerBlock - 1) / threadsPerBlock;

      // int sharedMemBytes = f2 * sizeof(float);

      // printf("blocksPerGrid = %d, threadsPerBlock = %d, sharedMemBytes = %d\n", blocksPerGrid, threadsPerBlock, sharedMemBytes);
      GPU_4loop_parK<<<blocksPerGrid, threadsPerBlock>>>(
        d_mode_0_ptr, d_mode_0_idx,
        d_mode_1_ptr, d_mode_1_idx,
        d_mode_2_ptr, d_mode_2_idx,
        d_values, d_arr_A, d_arr_B, d_arr_O, l, m, n, f1, f2, contraction,
        size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr,
        size_mode_0_idx, size_mode_1_idx, size_mode_2_idx, buffer_for_contraction_0_1
      );
      hipDeviceSynchronize();
      blocksPerGrid = (size_mode_1_idx + threadsPerBlock - 1) / threadsPerBlock;
      GPU_4loop_parJ<<<blocksPerGrid, threadsPerBlock>>>(
        d_mode_0_ptr, d_mode_0_idx,
        d_mode_1_ptr, d_mode_1_idx,
        d_mode_2_ptr, d_mode_2_idx,
        d_values, d_arr_A, d_arr_B, d_arr_O, l, m, n, f1, f2, contraction,
        size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr,
        size_mode_0_idx, size_mode_1_idx, size_mode_2_idx, buffer_for_contraction_0_1
      );

    }
     
    hipDeviceSynchronize();
  
    // Copy results back to host
    hipMemcpy(arr_O, d_arr_O, sizeof(float) * arr_O_size, hipMemcpyDeviceToHost);
  
    // Free device memory
    hipFree(d_mode_0_ptr);
    hipFree(d_mode_0_idx);
    hipFree(d_mode_1_ptr);
    hipFree(d_mode_1_idx);
    hipFree(d_mode_2_ptr);
    hipFree(d_mode_2_idx);
    hipFree(d_values);
    hipFree(d_arr_A);
    hipFree(d_arr_B);
    hipFree(d_arr_O);
  
  
  }
/*End of host function for GPU 4 loop Method using STREAMS*/
////////////////////////////////////////////////////////////////////

// Include the reference implementation for validation
#define INCLUDED_AS_LIBRARY
#include "v2_cpu_4loop.cu"

int main(int argc, char* argv[]) {
  bool verbose = false;
  string csf_file;
  uint64_t rank1 = 30, rank2 = 30;
  int ncm = 0;
  bool verify = false;  // Default: don't verify results
  
  // Parse command line arguments
  for (int i = 1; i < argc; i++) {
    string arg = argv[i];
    if (arg == "-v" || arg == "--verbose") {
      verbose = true;
    } else if (arg == "-r1" && i + 1 < argc) {
      rank1 = atoi(argv[++i]);
    } else if (arg == "-r2" && i + 1 < argc) {
      rank2 = atoi(argv[++i]);
    } else if ((arg == "-n" || arg == "--ncm") && i + 1 < argc) {
      ncm = atoi(argv[++i]);
    } else if (arg == "--verify") {
      verify = true;
    } else if (csf_file.empty()) {
      csf_file = arg;
    }
  }
  
  if (csf_file.empty()) {
    cerr << "Usage: " << argv[0] << " [options] <csf_file>" << endl;
    cerr << "Options:" << endl;
    cerr << "  -v, --verbose      Enable verbose output" << endl;
    cerr << "  -r1 <rank>         Set first factor matrix rank (default 30)" << endl;
    cerr << "  -r2 <rank>         Set second factor matrix rank (default 30)" << endl;
    cerr << "  -n, --ncm <mode>   Set contraction mode (0, 1, or 2, default 0)" << endl;
    cerr << "  --verify           Verify results against reference implementation" << endl;
    return 1;
  }
  
  try {
    // Load the CSF tensor
    CSFTensor tensor = readCSFTensor(csf_file);
    
    if (verbose) {
      cout << "Loaded tensor from " << csf_file << endl;
      cout << "Tensor dimensions: " << tensor.dimensions[0] << " x " << tensor.dimensions[1] << " x " << tensor.dimensions[2] << endl;
      cout << "Nonzeros: " << tensor.values.size() << endl;
    }
    
    // Convert CSF tensor to arrays
    uint64_t *mode_0_ptr, *mode_0_idx;
    uint64_t *mode_1_ptr, *mode_1_idx;
    uint64_t *mode_2_ptr, *mode_2_idx;
    float *values;
    int order;
    
    size_t size_mode_0_ptr = tensor.ptrs[0].size();
    size_t size_mode_1_ptr = tensor.ptrs[1].size();
    size_t size_mode_2_ptr = tensor.ptrs[2].size();
    size_t size_mode_0_idx = tensor.idxs[0].size();
    size_t size_mode_1_idx = tensor.idxs[1].size();
    size_t size_mode_2_idx = tensor.idxs[2].size();
    size_t total_values = tensor.values.size();

    if(verbose){
      cout << "size_mode_0_ptr: " << size_mode_0_ptr << endl;
      cout << "size_mode_1_ptr: " << size_mode_1_ptr << endl;
      cout << "size_mode_2_ptr: " << size_mode_2_ptr << endl;
      cout << "size_mode_0_idx: " << size_mode_0_idx << endl;
      cout << "size_mode_1_idx: " << size_mode_1_idx << endl;
      cout << "size_mode_2_idx: " << size_mode_2_idx << endl;
      cout << "total_values: " << total_values << endl;
    }
    
    vector<uint64_t> dimensions(tensor.order);
    for(int i = 0; i < tensor.order; i++){
      dimensions[i] = tensor.dimensions[i];
    }

    getCSFArrays(tensor, &mode_0_ptr, &mode_0_idx, 
                &mode_1_ptr, &mode_1_idx, 
                &mode_2_ptr, &mode_2_idx, 
                &values, &order);
    
    // Calculate matrix dimensions based on contraction mode
    uint64_t matrix_dim1 = getMatrixDim1(dimensions, ncm);
    uint64_t matrix_dim2 = getMatrixDim2(dimensions, ncm);
    uint64_t out_dim1 = getOutputDim1(dimensions, ncm);
    
    // Generate factor matrices
    float *arr_A = nullptr, *arr_B = nullptr;
    generate_matrix(matrix_dim1, rank1, 42, arr_A);
    generate_matrix(matrix_dim2, rank2, 43, arr_B);
    
    // Prepare output matrix dimensions
    uint64_t out_dim2 = rank1 * rank2;
    uint64_t arr_A_size = matrix_dim1 * rank1;
    uint64_t arr_B_size = matrix_dim2 * rank2;
    uint64_t arr_O_size = out_dim1 * out_dim2;
    
    if (verbose) {
      cout << "Matrix A dimensions: " << matrix_dim1 << " x " << rank1 << endl;
      cout << "Matrix B dimensions: " << matrix_dim2 << " x " << rank2 << endl;
      cout << "Output dimensions: " << out_dim1 << " x " << out_dim2 << endl;
    }
    
    // Allocate output array
    float* arr_O = allocate_aligned_array(arr_O_size);
    float* ref_O = nullptr;
    
    if (verify) {
      // Only allocate reference array if verification is needed
      ref_O = allocate_aligned_array(arr_O_size);
    }
    
    // Run this implementation (GPU 4-loop with each thread picking a k(non-zero)) first
    if (verbose) {
      cout << "Running GPU 4-loop(each thread picks a k(non-zero)) implementation..." << endl;
    }
    auto start = std::chrono::high_resolution_clock::now();
    
    
    GPU_4loop_parK_host_func(
      mode_0_ptr, mode_0_idx,
      mode_1_ptr, mode_1_idx,
      mode_2_ptr, mode_2_idx,
      values, arr_A, arr_B, arr_O,
      arr_A_size, arr_B_size, arr_O_size,
      ncm, dimensions[0], dimensions[1], dimensions[2], rank1, rank2,
      total_values,
      size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr,
      size_mode_0_idx, size_mode_1_idx, size_mode_2_idx
    );
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    
    bool valid = true;
    float ref_duration = 0.0;
    
    if (verify) {
        // Only run reference implementation and validate if requested
        if (verbose) {
          cout << "Running reference implementation (CPU 4-loop)..." << endl;
        }
        auto ref_start = std::chrono::high_resolution_clock::now();
        
        performContraction_cpu_2(
          mode_0_ptr, mode_0_idx,
          mode_1_ptr, mode_1_idx,
          mode_2_ptr, mode_2_idx,
          values, arr_A, arr_B, ref_O,
          arr_A_size, arr_B_size, arr_O_size, ncm,
          dimensions[0], dimensions[1], dimensions[2], rank1, rank2
        );
        
        auto ref_end = std::chrono::high_resolution_clock::now();
        ref_duration = std::chrono::duration_cast<std::chrono::microseconds>(ref_end - ref_start).count();
        
        // Validate results using compare_results from matrix_utils.h
        valid = compare_results(arr_O, ref_O, arr_O_size);
    }
    
    // Report results
    if (verbose) {
        cout << "GPU_1nz_per_thread (each thread picks a non-zero(k)) execution time: " << duration / 1000.0 << " ms" << endl;
        if (verify) {
          cout << "Reference execution time: " << ref_duration / 1000.0 << " ms" << endl;
          cout << "Speedup over reference: " << (float)ref_duration / duration << "x" << endl;
          cout << "Result validation: " << (valid ? "PASSED" : "FAILED") << endl;
        }
    } else {
        if (verify) {
          cout << "Method: GPU_1nz_per_thread , Time: " << duration / 1000.0 << " ms, Validation: " << (valid ? "PASSED" : "FAILED") << endl;
        } else {
          cout << "Method: GPU_1nz_per_thread , Time: " << duration / 1000.0 << " ms" << endl;
        }
    }
    
    // Clean up
    delete[] mode_0_ptr;
    delete[] mode_0_idx;
    delete[] mode_1_ptr;
    delete[] mode_1_idx;
    delete[] mode_2_ptr;
    delete[] mode_2_idx;
    delete[] values;
    delete[] arr_A;
    delete[] arr_B;
    free(arr_O);
    if (ref_O) free(ref_O);
    
    // return valid ? 0 : 1;
  }
  catch (const std::exception& e) {
      cerr << "Error: " << e.what() << endl;
      return 1;
  }
}