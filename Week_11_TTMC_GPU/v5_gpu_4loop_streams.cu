#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstring>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "csf_tensor.h"
#include "matrix_utils.h"

using namespace std;

// Helper macro for checking CUDA errors
#define cudaCheckError(call)                                                        \
do {                                                                                 \
    hipError_t err = call;                                                          \
    if (err != hipSuccess) {                                                        \
        std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << " -> "        \
                  << hipGetErrorString(err) << " (" << err << ")" << std::endl;      \
        std::exit(EXIT_FAILURE);                                                     \
    }                                                                                \
} while (0)

/////////////////////////////////////////////////////////////////////
/*Start of device function for GPU 4 loop Method using STREAMS*/
__global__ void GPU_4loop_streams(
  // uint64_t* mode_1_ptr,
  uint64_t* mode_1_idx,
  uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
  double* values, double* arr_A, double* arr_B,  
  double* arr_O, uint64_t l, uint64_t m, uint64_t n, uint64_t f1, uint64_t f2, int ncm,
  int size_mode_0_ptr, int size_mode_1_ptr, int size_mode_2_ptr,
  int size_mode_0_idx, int size_mode_1_idx, int size_mode_2_idx, uint64_t i, uint64_t j_ptr_offset
)
{
  extern __shared__ double buf[];
  uint64_t j, j_ptr, k, k_ptr, k_ptr_offset, index_A, index_B, index_O ;
  int r, s, r_offset, s_offset, WARP_SIZE = 32;
  double value, A_val;
  unsigned mask;

  j_ptr = j_ptr_offset + blockIdx.x;
  j = mode_1_idx[j_ptr];
  // uint64_t nnz_k = mode_2_ptr[j_ptr+1] - mode_2_ptr[j_ptr];
  
  int buf_index = threadIdx.y * blockDim.x + threadIdx.x;

  //NOTE; WORKS ONLY IF f2 < 1024
  if(buf_index < f2){
    buf[buf_index] = 0.0;
  }
  __syncthreads();
  
  // parallelize s across warps
  // block dimesion is 32 x 32. 
  // hence, each row of thread block will form a warp 
  // each row of thread block(a warp) picks a k, thus a nonzero of input tensor
  for(k_ptr_offset = mode_2_ptr[j_ptr]; k_ptr_offset < mode_2_ptr[j_ptr + 1]; k_ptr_offset += blockDim.x){
    k_ptr =  k_ptr_offset + threadIdx.x;
    if(k_ptr < mode_2_ptr[j_ptr + 1]){
      
      value = values[k_ptr];
      k = mode_2_idx[k_ptr];
      
      //Each thread in a warp picks a 's'
      for(s_offset = 0; s_offset < f2; s_offset += blockDim.y){ 
        s = s_offset + threadIdx.y;
        if(s < f2){
          mask = __activemask();
          index_B = k * f2 + s;
          double prod_val = value * arr_B[index_B];

          for(int shuffle_offset = WARP_SIZE/2; shuffle_offset > 0; shuffle_offset>>=1){
            prod_val += __shfl_down_sync(mask, prod_val, shuffle_offset);
          }
          if(threadIdx.x == 0) buf[s] += prod_val;
        //   atomicAdd(&buf[s], value * arr_B[index_B] );
        }
      }
    }
  }
  __syncthreads();
  
  //////////////////////////////////////////////////////////////////////////////////
  // parallelize 'r' across warps
  // block dimesion is 32 x 32. 
  // hence, each row of thread block will form a warp 
  // each row of thread block(a warp) picks a 'r'
  if(ncm == 0){
    for(r_offset = 0; r_offset < f1; r_offset += blockDim.y){
      r = r_offset + threadIdx.y;
      if(r < f1){
        index_A = j * f1 + r;
        A_val = arr_A[index_A];
        //Each thread in a warp picks a 's'
        for(s_offset = 0; s_offset < f2; s_offset += blockDim.x){
          s = s_offset + threadIdx.x;
          if(s < f2){
            index_O = i * f1 * f2 + r * f2  + s;
            //atomic add is required since different threadblocks in the same stream has same i
            atomicAdd(&arr_O[index_O], buf[s] * A_val);
          }
        }
        
      }
    }
  }
  else if(ncm == 1){
    for(r_offset = 0; r_offset < f1; r_offset += blockDim.y){
      r = r_offset + threadIdx.y;
      if(r < f1){
        index_A = i * f1 + r;
        A_val = arr_A[index_A];
        //Each thread in a warp picks a 's'
        for(s_offset = 0; s_offset < f2; s_offset += blockDim.x){
          s = s_offset + threadIdx.x;
          if(s < f2){
            index_O = j * f1 * f2 + r * f2  + s;
            //atomic add is required since different threadblocks in the same stream has same i
            atomicAdd(&arr_O[index_O], buf[s] * A_val);
          }
        }
        
      }
    }
  }
  // __syncthreads();
}

__global__ void GPU_4loop_streams_ncm_2_part_1(
  // uint64_t* mode_1_ptr,
  uint64_t* mode_1_idx,
  uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
  double* values, double* arr_A, double* arr_B,  
  double* arr_O, uint64_t l, uint64_t m, uint64_t n, uint64_t f1, uint64_t f2, int ncm,
  int size_mode_0_ptr, int size_mode_1_ptr, int size_mode_2_ptr,
  int size_mode_0_idx, int size_mode_1_idx, int size_mode_2_idx, uint64_t i, uint64_t j_ptr_offset,
  double* buffer_for_ncm_2, bool* k_index_buffer
)
{ 
  //shared memory will not be enough for 2d dense buf[k,s] of type double
  // for e.g. dim_k = 1024, dim_s = 32, the required memory is 32*8*1024 = 256kb
  uint64_t j, j_ptr, k, k_ptr, k_ptr_offset, index_B ;
  int  s, s_offset, buf_index;// WARP_SIZE = 32;
  double value;
  // unsigned mask;

  j_ptr = j_ptr_offset + blockIdx.x;
  j = mode_1_idx[j_ptr];
  
  // parallelize s across warps
  // block dimesion is 32 x 32. 
  // hence, each row of thread block will form a warp 
  // each column of thread block(a warp) picks a k, thus a nonzero of input tensor
  for(k_ptr_offset = mode_2_ptr[j_ptr]; k_ptr_offset < mode_2_ptr[j_ptr + 1]; k_ptr_offset += blockDim.x){
    k_ptr =  k_ptr_offset + threadIdx.x;
    if(k_ptr < mode_2_ptr[j_ptr + 1]){
      
      value = values[k_ptr];
      k = mode_2_idx[k_ptr];
      //since each column in threadblock has same k, threads from first column is enough to note that k
      if(threadIdx.y == 0) k_index_buffer[k] = true;
      
      //Each thread in a warp picks a 's'
      for(s_offset = 0; s_offset < f2; s_offset += blockDim.y){
        s = s_offset + threadIdx.y;
        if(s < f2){
          index_B = j * f2 + s;
          buf_index = k * f2 + s;
          double prod_val = value * arr_B[index_B];
          
          //warp shuffle cannot be used here because either k or s is changing along the both block dimension
          // mask = __activemask();
          // for(int shuffle_offset = WARP_SIZE/2; shuffle_offset > 0; shuffle_offset>>=1){
          //   prod_val += __shfl_down_sync(mask, prod_val, shuffle_offset);
          // }
          // if(threadIdx.x == 0) 
          atomicAdd(&buffer_for_ncm_2[buf_index], prod_val);
        }
      }
    }
  }
  // __syncthreads(); won't work because synchronization across blocks is required 
  
  //////////////////////////////////////////////////////////////////////////////////
  
}

// __global__ void pick_non_zero_Ks(bool* k_index_buffer, uint64_t* output_indices, int N) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= N) return;
//     if(threadIdx.x == 0 && blockIdx.x == 0){
//       __shared__ uint64_t counter = 0;
//     }
//     __syncthreads();

//     if (k_index_buffer[idx]) {
//       // Atomically get next free slot
//       int out_idx = atomicAdd(counter, 1);
//       output_indices[out_idx] = idx;
//     }
// }

__global__ void GPU_4loop_streams_ncm_2_part_2(
  uint64_t* mode_1_idx,
  uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
  double* values, double* arr_A, double* arr_B,  
  double* arr_O, uint64_t l, uint64_t m, uint64_t n, uint64_t f1, uint64_t f2, int ncm,
  int size_mode_0_ptr, int size_mode_1_ptr, int size_mode_2_ptr,
  int size_mode_0_idx, int size_mode_1_idx, int size_mode_2_idx, uint64_t i, uint64_t j_ptr_offset,
  double* buffer_for_ncm_2, bool* k_index_buffer
)
{
  uint64_t  k,  index_A, index_O ;
  int r, s, r_offset, s_offset, buf_index;
  double  A_val;
  k = blockIdx.x;
  if(k_index_buffer[k]){
    // parallelize 'r' across warps
    // block dimesion is 32 x 32. 
    // hence, each row of thread block will form a warp 
    // each row of thread block(a warp) picks a 'r'

    // if(threadIdx.x == 0 && threadIdx.y == 0) printf("k = %d", k);
    for(r_offset = 0; r_offset < f1; r_offset += blockDim.y){
      r = r_offset + threadIdx.y;
      if(r < f1){
        index_A = i * f1 + r;
        A_val = arr_A[index_A];
        //Each thread in a warp picks a 's'
        for(s_offset = 0; s_offset < f2; s_offset += blockDim.x){
          s = s_offset + threadIdx.x;
          if(s < f2){
            index_O = k * f1 * f2 + r * f2  + s;
            buf_index = k * f2 + s;
            //atomic add is required since different threadblocks in the same stream has same i
            atomicAdd(&arr_O[index_O], buffer_for_ncm_2[buf_index] * A_val);
          }
        }
      }
    }
  }
}
/*End of device function for GPU 4 loop Method using STREAMS*/
/////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////
/*Start of host function for GPU 4 loop Method using STREAMS*/
void GPU_4loop_host_func(
  uint64_t* mode_0_ptr, uint64_t* mode_0_idx,
  uint64_t* mode_1_ptr, uint64_t* mode_1_idx,
  uint64_t* mode_2_ptr, uint64_t* mode_2_idx,
  double* values, double* arr_A, double* arr_B,  
  double* arr_O, uint64_t arr_A_size, uint64_t arr_B_size, uint64_t arr_O_size, int contraction, 
  uint64_t l, uint64_t m, uint64_t n, uint64_t f1, uint64_t f2, uint64_t total_values,
  int size_mode_0_ptr, int size_mode_1_ptr, int size_mode_2_ptr,
  int size_mode_0_idx, int size_mode_1_idx, int size_mode_2_idx)
  {
    // Allocate device memory
    // uint64_t *d_mode_0_ptr, *d_mode_0_idx, *d_mode_1_ptr;
    uint64_t *d_mode_1_idx, *d_mode_2_ptr, *d_mode_2_idx;
    double *d_values, *d_arr_A, *d_arr_B, *d_arr_O;
    // double* buffer_for_contraction_0_1;
    // double* buffer_for_contraction_2;
    // int* k_buffer_for_contraction_2;
  
    // hipMalloc(&d_mode_0_ptr, sizeof(uint64_t) * size_mode_0_ptr);
    // hipMalloc(&d_mode_0_idx, sizeof(uint64_t) * size_mode_0_idx);
    // hipMalloc(&d_mode_1_ptr, sizeof(uint64_t) * size_mode_1_ptr);
    hipMalloc(&d_mode_1_idx, sizeof(uint64_t) * size_mode_1_idx);
    hipMalloc(&d_mode_2_ptr, sizeof(uint64_t) * size_mode_2_ptr);
    hipMalloc(&d_mode_2_idx, sizeof(uint64_t) * size_mode_2_idx);
    hipMalloc(&d_values, sizeof(double) * total_values);
    hipMalloc(&d_arr_A, sizeof(double) * arr_A_size);
    hipMalloc(&d_arr_B, sizeof(double) * arr_B_size);
    hipMalloc(&d_arr_O, sizeof(double) * arr_O_size);
  
  
    // // parallelising 'j_ptr' for contraction = 0 and contraction = 1 :
    // hipMalloc(&buffer_for_contraction_0_1, f2 * size_mode_1_idx * sizeof(double));
  
    // // parallelising 'j_ptr' for contraction = 2 :
    // hipMalloc(&buffer_for_contraction_2, n * f2 * size_mode_1_idx * sizeof(double));
    // hipMalloc(&k_buffer_for_contraction_2, n * size_mode_1_idx * sizeof(int));
  
    // Copy data to device
    // hipMemcpy(d_mode_0_ptr, mode_0_ptr, sizeof(uint64_t) * size_mode_0_ptr, hipMemcpyHostToDevice);
    // hipMemcpy(d_mode_0_idx, mode_0_idx, sizeof(uint64_t) * size_mode_0_idx, hipMemcpyHostToDevice);
    // hipMemcpy(d_mode_1_ptr, mode_1_ptr, sizeof(uint64_t) * size_mode_1_ptr, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_1_idx, mode_1_idx, sizeof(uint64_t) * size_mode_1_idx, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_2_ptr, mode_2_ptr, sizeof(uint64_t) * size_mode_2_ptr, hipMemcpyHostToDevice);
    hipMemcpy(d_mode_2_idx, mode_2_idx, sizeof(uint64_t) * size_mode_2_idx, hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, sizeof(double) * total_values, hipMemcpyHostToDevice);
    hipMemcpy(d_arr_A, arr_A, sizeof(double) * arr_A_size, hipMemcpyHostToDevice);
    hipMemcpy(d_arr_B, arr_B, sizeof(double) * arr_B_size, hipMemcpyHostToDevice);
    hipMemcpy(d_arr_O, arr_O, sizeof(double) * arr_O_size, hipMemcpyHostToDevice);
    
    
    // // parallelising 'j_ptr' for contraction = 0 and contraction = 1 :
    // hipMemset(buffer_for_contraction_0_1, 0, f2 * size_mode_1_idx * sizeof(double));
    
    
    // // parallelising 'j_ptr' for contraction = 2 :
    // hipMemset(buffer_for_contraction_2, 0, n * f2 * size_mode_1_idx * sizeof(double));
    // hipMemset(k_buffer_for_contraction_2, 0, n * size_mode_1_idx * sizeof(int));
    
    // Stream setup
    uint64_t i, itr, j_ptr_offset;
    // uint64_t NUM_STREAMS = size_mode_0_idx;
    uint64_t NUM_STREAMS = 4; //increasing beyond 4 doesn't improve performance
    
    hipStream_t streams[NUM_STREAMS];
    for (itr = 0; itr < NUM_STREAMS; ++itr) {
      hipStreamCreate(&streams[itr]);
    }
    
    // uint64_t mode_1_idx_offset, mode_2_ptr_offset, mode_2_idx_offset, mode_1_idx_num_elements;
    // Launch kernels
    if (contraction == 0 || contraction == 1) {
      cout << "No. of streams = " << NUM_STREAMS <<endl;
      for (uint64_t i_ptr = 0; i_ptr < mode_0_ptr[1]; ++i_ptr) {
        i = mode_0_idx[i_ptr];
        j_ptr_offset = mode_1_ptr[i_ptr];
        
        // int blocksPerGrid = mode_1_ptr[i_ptr + 1] - mode_1_ptr[i_ptr];
        dim3 gridDim(mode_1_ptr[i_ptr + 1] - mode_1_ptr[i_ptr]);
        dim3 blockDim(32, 32);
        int sharedMemBytes = f2 * sizeof(double);
        
        // mode_1_idx_offset = mode_1_ptr[i_ptr] ;
        // mode_1_idx_num_elements = mode_1_ptr[i_ptr + 1] - mode_1_ptr[i_ptr];
        // mode_2_ptr_offset = mode_2
        // mode_2_idx_offset;
        // hipMemcpyAsync(d_mode_1_idx + mode_1_idx_offset, mode_1_idx + mode_1_idx_offset, sizeof(uint64_t) * mode_1_idx_num_elements, hipMemcpyHostToDevice, streams[i_ptr%NUM_STREAMS]);
        // hipMemcpyAsync(d_mode_2_ptr + mode_2_ptr_offset, mode_2_ptr + mode_2_ptr_offset, sizeof(uint64_t) * mode_2_ptr_num_elements, hipMemcpyHostToDevice);
        // hipMemcpyAsync(d_mode_2_idx + mode_2_idx_offset, mode_2_idx + mode_2_idx_offset, sizeof(uint64_t) * mode_2_idx_num_elememts, hipMemcpyHostToDevice);
        // hipMemcpyAsync(d_values + mode_2_idx_offset, values + mode_2_idx_offset, sizeof(double) * mode_2_idx_num_elememts, hipMemcpyHostToDevice);
        
        //TO-DO: Instead, use hipStreamQuery to find idle streams and then assign work. will it improve performance? No I think
        GPU_4loop_streams<<<gridDim, blockDim, sharedMemBytes, streams[i_ptr%NUM_STREAMS]>>>(
          // d_mode_1_ptr, 
          d_mode_1_idx, d_mode_2_ptr, d_mode_2_idx,
          d_values, d_arr_A, d_arr_B, d_arr_O, l, m, n, f1, f2, contraction,
          size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr,
          size_mode_0_idx, size_mode_1_idx, size_mode_2_idx,
          i, j_ptr_offset
        );
        hipGetLastError();  // Check launch err;
      }
    }
    else if(contraction == 2){
      double* buffer_for_ncm_2;
      bool* k_index_buffer;
      
      NUM_STREAMS = 1;
      cout << "No. of streams = " << NUM_STREAMS <<endl;

      hipMalloc(&buffer_for_ncm_2, n * f2 * NUM_STREAMS * sizeof(double));
      hipMalloc(&k_index_buffer, n * NUM_STREAMS * sizeof(bool));
      
      // hipMalloc(&k_indices, n * NUM_STREAMS * sizeof(uint64_t));
      // hipMalloc(&counter,  NUM_STREAMS * sizeof(uint64_t));
      
      // hipMemset(buffer_for_ncm_2 , 0, n * f2  * NUM_STREAMS * sizeof(double));
      // hipMemset(k_index_buffer, 0, n  * NUM_STREAMS * sizeof(bool));

      
      for (uint64_t i_ptr = 0; i_ptr < mode_0_ptr[1]; ++i_ptr) {
        i = mode_0_idx[i_ptr];
        j_ptr_offset = mode_1_ptr[i_ptr];
        
        hipMemset(buffer_for_ncm_2 + n * f2 * (i_ptr % NUM_STREAMS), 0, n * f2  * sizeof(double));
        hipMemset(k_index_buffer + n * (i_ptr % NUM_STREAMS), 0, n  * sizeof(bool));
        
        dim3 gridDim(mode_1_ptr[i_ptr + 1] - mode_1_ptr[i_ptr]);
        dim3 blockDim(32, 32);

        GPU_4loop_streams_ncm_2_part_1<<<gridDim, blockDim, 0, streams[i_ptr%NUM_STREAMS]>>>(
          d_mode_1_idx, d_mode_2_ptr, d_mode_2_idx,
          d_values, d_arr_A, d_arr_B, d_arr_O, l, m, n, f1, f2, contraction,
          size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr,
          size_mode_0_idx, size_mode_1_idx, size_mode_2_idx,
          i, j_ptr_offset, buffer_for_ncm_2 + n * f2 * (i_ptr % NUM_STREAMS), k_index_buffer + n * (i_ptr % NUM_STREAMS)
        );

        // hipDeviceSynchronize();
        // pick_non_zero_Ks(k_index_buffer + n * (i_ptr % NUM_STREAMS), k_indices + n * (i_ptr % NUM_STREAMS),  n)

        gridDim.x = n; //TO-DO: have to be optimized
        GPU_4loop_streams_ncm_2_part_2<<<gridDim, blockDim, 0, streams[i_ptr%NUM_STREAMS]>>>(
          d_mode_1_idx, d_mode_2_ptr, d_mode_2_idx,
          d_values, d_arr_A, d_arr_B, d_arr_O, l, m, n, f1, f2, contraction,
          size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr,
          size_mode_0_idx, size_mode_1_idx, size_mode_2_idx,
          i, j_ptr_offset, buffer_for_ncm_2 + n * (i_ptr % NUM_STREAMS), k_index_buffer + n * (i_ptr % NUM_STREAMS)
        );
        hipGetLastError();  // Check launch err;
        // hipStreamSynchronize(streams[i_ptr % NUM_STREAMS]);
      }
      
    }

  // Sync and destroy streams
  for ( itr = 0; itr < NUM_STREAMS; ++itr) {
    hipStreamSynchronize(streams[itr]);
    hipStreamDestroy(streams[itr]);
  }

  
    // Copy results back to host
    hipMemcpy(arr_O, d_arr_O, sizeof(double) * arr_O_size, hipMemcpyDeviceToHost);
  
    // Free device memory
    // hipFree(d_mode_0_ptr);
    // hipFree(d_mode_0_idx);
    // hipFree(d_mode_1_ptr);
    hipFree(d_mode_1_idx);
    hipFree(d_mode_2_ptr);
    hipFree(d_mode_2_idx);
    hipFree(d_values);
    hipFree(d_arr_A);
    hipFree(d_arr_B);
    hipFree(d_arr_O);
  
    // hipFree(buffer_for_contraction_0_1);
    // hipFree(buffer_for_contraction_2);
    // hipFree(k_buffer_for_contraction_2);
  }
/*End of host function for GPU 4 loop Method using STREAMS*/
////////////////////////////////////////////////////////////////////

// Include the reference implementation for validation
#define INCLUDED_AS_LIBRARY
#include "v2_cpu_4loop.cu"

int main(int argc, char* argv[]) {
    bool verbose = false;
    string csf_file;
    uint64_t rank1 = 30, rank2 = 30;
    int ncm = 0;
    
    // Parse command line arguments
    for (int i = 1; i < argc; i++) {
        string arg = argv[i];
        if (arg == "-v" || arg == "--verbose") {
            verbose = true;
        } else if (arg == "-r1" && i + 1 < argc) {
            rank1 = atoi(argv[++i]);
        } else if (arg == "-r2" && i + 1 < argc) {
            rank2 = atoi(argv[++i]);
        } else if ((arg == "-n" || arg == "--ncm") && i + 1 < argc) {
            ncm = atoi(argv[++i]);
        } else if (csf_file.empty()) {
            csf_file = arg;
        }
    }
    
    if (csf_file.empty()) {
        cerr << "Usage: " << argv[0] << " [options] <csf_file>" << endl;
        cerr << "Options:" << endl;
        cerr << "  -v, --verbose      Enable verbose output" << endl;
        cerr << "  -r1 <rank>         Set first factor matrix rank (default 30)" << endl;
        cerr << "  -r2 <rank>         Set second factor matrix rank (default 30)" << endl;
        cerr << "  -n, --ncm <mode>   Set contraction mode (0, 1, or 2, default 0)" << endl;
        return 1;
    }
    
    try {
        // Load the CSF tensor
        CSFTensor tensor = readCSFTensor(csf_file);
        
        if (verbose) {
            cout << "Loaded tensor from " << csf_file << endl;
            cout << "Tensor dimensions: " << tensor.dimensions[0] << " x " << tensor.dimensions[1] << " x " << tensor.dimensions[2] << endl;
            cout << "Nonzeros: " << tensor.values.size() << endl;
        }
        
        // Convert CSF tensor to arrays
        uint64_t *mode_0_ptr, *mode_0_idx;
        uint64_t *mode_1_ptr, *mode_1_idx;
        uint64_t *mode_2_ptr, *mode_2_idx;
        double *values;
        int order;
        
        getCSFArrays(tensor, &mode_0_ptr, &mode_0_idx, 
                    &mode_1_ptr, &mode_1_idx, 
                    &mode_2_ptr, &mode_2_idx, 
                    &values, &order);
        
        size_t size_mode_0_ptr = tensor.ptrs[0].size();
        size_t size_mode_1_ptr = tensor.ptrs[1].size();
        size_t size_mode_2_ptr = tensor.ptrs[2].size();
        size_t size_mode_0_idx = tensor.idxs[0].size();
        size_t size_mode_1_idx = tensor.idxs[1].size();
        size_t size_mode_2_idx = tensor.idxs[2].size();
        size_t total_values = tensor.values.size();
        
        // Calculate matrix dimensions based on contraction mode
        uint64_t matrix_dim1 = getMatrixDim1(tensor.dimensions, ncm);
        uint64_t matrix_dim2 = getMatrixDim2(tensor.dimensions, ncm);
        uint64_t out_dim1 = getOutputDim1(tensor.dimensions, ncm);
        
        // Generate factor matrices
        double *arr_A = nullptr, *arr_B = nullptr;
        generate_matrix(matrix_dim1, rank1, 42, arr_A);
        generate_matrix(matrix_dim2, rank2, 43, arr_B);
        
        // Prepare output matrix dimensions
        uint64_t out_dim2 = rank1 * rank2;
        uint64_t arr_A_size = matrix_dim1 * rank1;
        uint64_t arr_B_size = matrix_dim2 * rank2;
        uint64_t arr_O_size = out_dim1 * out_dim2;
        
        if (verbose) {
            cout << "Matrix A dimensions: " << matrix_dim1 << " x " << rank1 << endl;
            cout << "Matrix B dimensions: " << matrix_dim2 << " x " << rank2 << endl;
            cout << "Output dimensions: " << out_dim1 << " x " << out_dim2 << endl;
        }
        
        // Allocate output arrays
        double* arr_O = allocate_aligned_array(arr_O_size);
        double* ref_O = allocate_aligned_array(arr_O_size);
        
        // Run this implementation (GPU 4-loop with streams) first
        if (verbose) {
            cout << "Running GPU 4-loop with streams implementation..." << endl;
        }
        auto start = std::chrono::high_resolution_clock::now();
        
        // pinned memory for streams
        size_t ptr_size_0 = sizeof(uint64_t) * size_mode_0_ptr;
        size_t idx_size_0 = sizeof(uint64_t) * size_mode_0_idx;
        size_t ptr_size_1 = sizeof(uint64_t) * size_mode_1_ptr;
        size_t idx_size_1 = sizeof(uint64_t) * size_mode_1_idx;
        size_t ptr_size_2 = sizeof(uint64_t) * size_mode_2_ptr;
        size_t idx_size_2 = sizeof(uint64_t) * size_mode_2_idx;
        size_t val_size   = sizeof(double)   * total_values;

        // Register host memory
        hipHostRegister(mode_0_ptr, ptr_size_0, hipHostRegisterDefault);
        hipHostRegister(mode_0_idx, idx_size_0, hipHostRegisterDefault);
        hipHostRegister(mode_1_ptr, ptr_size_1, hipHostRegisterDefault);
        hipHostRegister(mode_1_idx, idx_size_1, hipHostRegisterDefault);
        hipHostRegister(mode_2_ptr, ptr_size_2, hipHostRegisterDefault);
        hipHostRegister(mode_2_idx, idx_size_2, hipHostRegisterDefault);
        hipHostRegister(values,     val_size,   hipHostRegisterDefault);

        GPU_4loop_host_func(
            mode_0_ptr, mode_0_idx,
            mode_1_ptr, mode_1_idx,
            mode_2_ptr, mode_2_idx,
            values, arr_A, arr_B, arr_O,
            arr_A_size, arr_B_size, arr_O_size,
            ncm, tensor.dimensions[0], tensor.dimensions[1], tensor.dimensions[2], rank1, rank2,
            tensor.values.size(),
            size_mode_0_ptr, size_mode_1_ptr, size_mode_2_ptr,
            size_mode_0_idx, size_mode_1_idx, size_mode_2_idx
        );
        
        hipHostUnregister(mode_0_ptr);
        hipHostUnregister(mode_0_idx);
        hipHostUnregister(mode_1_ptr);
        hipHostUnregister(mode_1_idx);
        hipHostUnregister(mode_2_ptr);
        hipHostUnregister(mode_2_idx);
        hipHostUnregister(values);
        
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
        
        // Now run reference implementation (CPU 4-loop) for validation
        if (verbose) {
            cout << "Running reference implementation (CPU 4-loop)..." << endl;
        }
        auto ref_start = std::chrono::high_resolution_clock::now();
        
        performContraction_cpu_2(
            mode_0_ptr, mode_0_idx,
            mode_1_ptr, mode_1_idx,
            mode_2_ptr, mode_2_idx,
            values, arr_A, arr_B, ref_O,
            arr_A_size, arr_B_size, arr_O_size, ncm,
            tensor.dimensions[0], tensor.dimensions[1], tensor.dimensions[2], rank1, rank2
        );
        
        auto ref_end = std::chrono::high_resolution_clock::now();
        auto ref_duration = std::chrono::duration_cast<std::chrono::microseconds>(ref_end - ref_start).count();
        
        // Validate results using compare_results from matrix_utils.h
        bool valid = compare_results(arr_O, ref_O, arr_O_size);
        
        // Report results
        if (verbose) {
            cout << "GPU 4-loop streams execution time: " << duration / 1000.0 << " ms" << endl;
            cout << "Reference execution time: " << ref_duration / 1000.0 << " ms" << endl;
            cout << "Speedup over reference: " << (double)ref_duration / duration << "x" << endl;
            cout << "Result validation: " << (valid ? "PASSED" : "FAILED") << endl;
        } else {
            cout << "Method: GPU 4-loop streams, Time: " << duration / 1000.0 << " ms, Validation: " << (valid ? "PASSED" : "FAILED") << endl;
        }
        
        // Clean up
        delete[] mode_0_ptr;
        delete[] mode_0_idx;
        delete[] mode_1_ptr;
        delete[] mode_1_idx;
        delete[] mode_2_ptr;
        delete[] mode_2_idx;
        delete[] values;
        delete[] arr_A;
        delete[] arr_B;
        free(arr_O);
        free(ref_O);
        
        return valid ? 0 : 1;
    }
    catch (const std::exception& e) {
        cerr << "Error: " << e.what() << endl;
        return 1;
    }
}