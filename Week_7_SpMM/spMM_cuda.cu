#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>   // For timing
#include <cstdlib>  // For atoi
#include "COOtoCSR.h"


__device__ inline double atomicAddDouble(double* address, double value) {
    unsigned long long* address_as_ulong = reinterpret_cast<unsigned long long*>(address);
    unsigned long long old = *address_as_ulong;
    unsigned long long assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ulong, assumed, __double_as_longlong(value + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


__global__ void spmmKernel(int64_t* row_pointers, int64_t* col_indices, double* values,
                           double* B, double* C, int64_t m, int64_t n, int64_t p) {
    int64_t row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m) {
        for (int64_t idx = row_pointers[row]; idx < row_pointers[row + 1]; ++idx) {
            int64_t col = col_indices[idx];
            double val = values[idx];
            for (int64_t j = 0; j < p; ++j) {
                atomicAddDouble(&C[row * p + j], val * B[col * p + j]);
            }
        }
    }
}


// Function to perform SpMM using CUDA
void spmm(int64_t m, int64_t n, int64_t p, int64_t* row_pointers, int64_t* col_indices, double* values, double* B, double* C) {
    // Device memory allocation
    int64_t *d_row_pointers, *d_col_indices;
    double *d_values, *d_B, *d_C;

    hipMalloc((void**)&d_row_pointers, (m + 1) * sizeof(int64_t));
    hipMalloc((void**)&d_col_indices, row_pointers[m] * sizeof(int64_t)); // Use the number of non-zeros
    hipMalloc((void**)&d_values, row_pointers[m] * sizeof(double)); // Use the number of non-zeros
    hipMalloc((void**)&d_B, n * p * sizeof(double));
    hipMalloc((void**)&d_C, m * p * sizeof(double));

    // Copy data to device
    hipMemcpy(d_row_pointers, row_pointers, (m + 1) * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, col_indices, row_pointers[m] * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, row_pointers[m] * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * p * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m * p * sizeof(double), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    int blockSize = 256;
    int gridSize = (m + blockSize - 1) / blockSize;

    auto start_gpu = std::chrono::high_resolution_clock::now();
    spmmKernel<<<gridSize, blockSize>>>(d_row_pointers, d_col_indices, d_values, d_B, d_C, m, n, p);
    auto end_gpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> gpu_time = end_gpu - start_gpu;

    std::cout << "Thread block size: " << blockSize << ", SpMM GPU execution time: " << gpu_time.count() << " seconds." << std::endl;

    // Copy result back to host
    hipMemcpy(C, d_C, m * p * sizeof(double), hipMemcpyDeviceToHost);

    // Display the result (C matrix)
    std::cout << "Result Matrix C:" << std::endl;
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < p; ++j) {
            std::cout << C[i * p + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_row_pointers);
    hipFree(d_col_indices);
    hipFree(d_values);
    hipFree(d_B);
    hipFree(d_C);
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <MatrixMarketFile> <Number of Columns in Matrix B>" << std::endl;
        return 1;
    }

    // Declare arrays to hold CSR data
    int64_t *row_pointers = nullptr;
    int64_t *col_indices = nullptr;
    double *values = nullptr;
    int64_t A_rows, A_cols, A_nonzeros;

    std::string filename = argv[1];
    int64_t p = std::stoll(argv[2]);
    convertToCSR(filename, row_pointers, col_indices, values, A_rows, A_cols, A_nonzeros);

    // Allocate and initialize dense matrix B with 1.0 (row-major format)
    double *B = new double[A_cols * p];
    for (int i = 0; i < A_cols; ++i) {
        for (int j = 0; j < p; ++j) {
            B[i * p + j] = (i == j) ? 1.0 : 0.0;
        }
    }

    // Allocate result matrix C and initialize to 0.0
    double *C = new double[A_rows * p]();
    
    // Perform SpMM
    spmm(A_rows, A_cols, p, row_pointers, col_indices, values, B, C);

    // Display CSR data
    // std::cout << "Values: ";
    // for (int i = 0; i < A_nonzeros; i++) std::cout << values[i] << " ";
    // std::cout << "\nColumn Indices: ";
    // for (int i = 0; i < A_nonzeros; i++) std::cout << col_indices[i] << " ";
    // std::cout << "\nRow Pointers: ";
    // for (int i = 0; i <= A_rows; i++) std::cout << row_pointers[i] << " ";
    // std::cout << std::endl;
    
    // Clean up dynamically allocated memory
    delete[] B;
    delete[] C;
    delete[] row_pointers;
    delete[] col_indices;
    delete[] values;

    return 0;
}
